
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <dirent.h>
#include <fcntl.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <climits>
using namespace std;

bool scan_dir(const char* dir, vector<string> &files)
{
    DIR *dp;                                            // 定义子目录流指针  
    struct dirent *entry;                               // 定义dirent结构指针保存后续目录  
    struct stat statbuf;                                // 定义statbuf结构保存文件属性  
    if((dp = opendir(dir)) == NULL) return false;       // 打开目录，获取子目录流指针，判断操作是否成功  
    chdir (dir);                                        // 切换到当前目录  
    while((entry = readdir(dp)) != NULL)                // 获取下一级目录信息，如果未否则循环  
    {
        lstat(entry->d_name, &statbuf);                 // 获取下一级成员属性  
        if(S_IFDIR &statbuf.st_mode)                    // 判断下一级成员是否是目录  
        {
            if (strcmp(".", entry->d_name) == 0 || strcmp("..", entry->d_name) == 0)
              continue;

            if (!scan_dir(entry->d_name, files)) return false;              // 递归调用自身，扫描下一级目录的内容  
        }
        else
        {
            files.push_back(string(getcwd(NULL, 0)) + "/" + string(entry->d_name));
        }
    }
    chdir("..");                                                  // 回到上级目录  
    closedir(dp);                                                 // 关闭子目录流  
    return true;
}

void read_data(vector<string> &files, vector<int> &data)
{
    char buff[256];
    for (int i=0; i<files.size(); i++)
    {
        ifstream fin(files[i].c_str(),ios::in);
        if (!fin.is_open())
        {
            cout<<"load "<<files[i]<<" fail."<<endl;
            continue;
        }
        else cout<<"loading "<<files[i]<<endl;
        while(fin.getline(buff,256))
            data.push_back(atoi(buff));
        fin.close();
    }
}

static void HandleError( hipError_t err, const char *file, int line ) 
{
    if (err != hipSuccess) 
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// a>=b return 0   a<b return 1
__device__ int cmp_without_if(int a, int b)
{
    int c = a - b;
    return (c >> 31) & 0x1;
}

__device__ int max_without_if(int a, int b)
{
    int c = a - b;
    int k = (c >> 31) & 0x1;
    return a - k * c;
}

__device__ int min_without_if(int a, int b)
{
    return (a + b) - max_without_if(a,b);
}

__global__ void reduce_sum(int *d_arr, int len, int *d_sum)
{
    __shared__ int cache[1024];
    int tid = threadIdx.x;

    cache[tid] = 0;
    for(int i=tid; i<len; i+=blockDim.x)
        cache[tid] += d_arr[i];
    __syncthreads();

    int i = blockDim.x/2;
    while(i)
    {
        if (tid < i) cache[tid] += cache[tid+i];
        __syncthreads();
        i /= 2;
    }
    if (tid == 0) *d_sum = cache[0];
}

__global__ void reduce_min_max(int *d_min_arr, int *d_max_arr, int len, int *d_min, int *d_max)
{
    __shared__ int mincache[1024];
    __shared__ int maxcache[1024];
    int tid = threadIdx.x;
    mincache[tid] = INT_MAX/2;
    maxcache[tid] = INT_MIN/2;

    for(int i=tid; i<len; i+=blockDim.x)
    {
        mincache[tid] = min_without_if(mincache[tid], d_min_arr[i]);
        maxcache[tid] = max_without_if(maxcache[tid], d_max_arr[i]); 
    }
    __syncthreads();

    int i = blockDim.x/2;
    while(i)
    {
        if (tid < i)
        {
            mincache[tid] = min_without_if(mincache[tid], mincache[tid+i]);
            maxcache[tid] = max_without_if(maxcache[tid], maxcache[tid+i]); 
        }
        __syncthreads();
        i /= 2;
    }
    if (tid == 0)
    {
        *d_min = mincache[0];
        *d_max = maxcache[0];
    }
}

__global__ void partition_min_max(int *d_arr, int *d_flag, int len, int *d_min_arr, int *d_max_arr)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int max = INT_MIN/2, min = INT_MAX/2;
    for (int i=tid; i<len; i+=blockDim.x*gridDim.x)
    {
        min = d_flag[i] * min_without_if(min, d_arr[i]) + (1 - d_flag[i]) * min;
        max = d_flag[i] * max_without_if(max, d_arr[i]) + (1 - d_flag[i]) * max;
    }
    d_min_arr[tid] = min;
    d_max_arr[tid] = max;
}

__global__ void partition_pivot(int *d_arr, int *d_flag, int len, int *d_pivot_arr)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    d_pivot_arr[tid] = INT_MAX/2;
    for (int i=tid; i<len; i+=blockDim.x*gridDim.x)
    {
        if (d_flag[i])
        {
            d_pivot_arr[tid] = d_arr[i];
            return;
        }
    }
}

__global__ void partition_less_num(int pivot, int *d_arr, int *d_flag, int len, int *d_lt_num_arr)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    d_lt_num_arr[tid] = 0;
    for (int i=tid; i<len; i+=blockDim.x*gridDim.x)
    {
        d_lt_num_arr[tid] += d_flag[i] * cmp_without_if(d_arr[i], pivot);
    }
}

__global__ void partition_remove_less(int pivot,int *d_arr,int *d_flag,int len)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i=tid; i<len; i+=blockDim.x*gridDim.x)
    {
        d_flag[i] = d_flag[i] * (1 - cmp_without_if(d_arr[i], pivot));
    }
}

__global__ void partition_remove_great(int pivot, int *d_arr, int *d_flag, int len)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i=tid; i<len; i+=blockDim.x*gridDim.x)
    {
        d_flag[i] = d_flag[i] * cmp_without_if(d_arr[i], pivot);
    }
}

__host__ int nth_order_stat(int target_order_stat, int *h_arr, int *d_arr, int *d_flag, int len);

__host__ int partition_on_pivot(int target_order_stat, int *h_arr, int *d_arr, int *d_flag, int len, int pivot_index)
{
    int threadNum = 1024;
    int blockNum = (len + threadNum - 1) / threadNum;
    blockNum = blockNum <= 1024 ? blockNum : 1024;

    static int pivot_arr[1024];
    int pivot;
    if (pivot_index == 0)
    {
        int *d_pivot_arr;
        HANDLE_ERROR( hipMalloc((void**)&d_pivot_arr,sizeof(int)*1024) );
        partition_pivot<<<1,1024>>>(d_arr,d_flag,len,d_pivot_arr);
        HANDLE_ERROR( hipMemcpy(&pivot_arr,d_pivot_arr,sizeof(int)*1024,hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipFree(d_pivot_arr) );
        pivot = pivot_arr[0];
    }
    else
        pivot = pivot_arr[rand()%1024];

    int *d_lt_num_arr, *d_lt_num, lt_num;
    HANDLE_ERROR( hipMalloc((void**)&d_lt_num_arr,sizeof(int)*blockNum*threadNum) );
    HANDLE_ERROR( hipMalloc((void**)&d_lt_num,sizeof(int)) );

    partition_less_num<<<blockNum,threadNum>>>(pivot,d_arr,d_flag,len,d_lt_num_arr);
    reduce_sum<<<1,1024>>>(d_lt_num_arr,blockNum*threadNum,d_lt_num);
    HANDLE_ERROR( hipMemcpy(&lt_num,d_lt_num,sizeof(int),hipMemcpyDeviceToHost) );

    HANDLE_ERROR( hipFree(d_lt_num_arr) );
    HANDLE_ERROR( hipFree(d_lt_num) );

    int pivot_order_stat = lt_num + 1;
    
    if (pivot_order_stat == target_order_stat) 
        return pivot;
    else if (pivot_order_stat == 1)
        return partition_on_pivot(target_order_stat, h_arr, d_arr, d_flag, len, pivot_index+1);
    else if (pivot_order_stat < target_order_stat)
    {
        target_order_stat -= lt_num;
        partition_remove_less<<<blockNum,threadNum>>>(pivot,d_arr,d_flag,len);
        return nth_order_stat(target_order_stat, h_arr, d_arr, d_flag, len);
    }
    else
    {
        partition_remove_great<<<blockNum,threadNum>>>(pivot,d_arr,d_flag,len);
        return nth_order_stat(target_order_stat, h_arr, d_arr, d_flag, len);
    }
}

__host__ int nth_order_stat(int target_order_stat, int *h_arr, int *d_arr, int *d_flag, int len)
{
    int threadNum = 1024;
    int blockNum = (len + threadNum - 1) / threadNum;
    blockNum = blockNum <= 1024 ? blockNum : 1024;

    int *d_min_arr, *d_max_arr;
    int h_min, h_max, *d_min, *d_max;
    HANDLE_ERROR( hipMalloc((void**)&d_min_arr,sizeof(int)*blockNum*threadNum) );
    HANDLE_ERROR( hipMalloc((void**)&d_max_arr,sizeof(int)*blockNum*threadNum) );  
    HANDLE_ERROR( hipMalloc((void**)&d_min,sizeof(int)) );  
    HANDLE_ERROR( hipMalloc((void**)&d_max,sizeof(int)) );  

    partition_min_max<<<blockNum,threadNum>>>(d_arr,d_flag,len,d_min_arr,d_max_arr);
    reduce_min_max<<<1,1024>>>(d_min_arr,d_max_arr,blockNum*threadNum,d_min,d_max);
    HANDLE_ERROR( hipMemcpy(&h_min,d_min,sizeof(int),hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy(&h_max,d_max,sizeof(int),hipMemcpyDeviceToHost) );

    HANDLE_ERROR (hipFree(d_min_arr));
    HANDLE_ERROR (hipFree(d_max_arr));
    HANDLE_ERROR (hipFree(d_min));
    HANDLE_ERROR (hipFree(d_max));

    if (h_max == h_min || target_order_stat == 1)
        return h_min;
    else if (target_order_stat == len)
        return h_max;
    else
        return partition_on_pivot(target_order_stat, h_arr, d_arr, d_flag, len, 0);
}

__host__ int median(int *arr, int len)
{
    int *d_arr;
    hipMalloc((void**)&d_arr,sizeof(int)*len);
    hipMemcpy(d_arr,arr,sizeof(int)*len,hipMemcpyHostToDevice);

    int *h_flag = new int[len];
    for (int i=0; i<len; i++) h_flag[i] = 1;

    int *d_flag;
    hipMalloc((void**)&d_flag,sizeof(int)*len);
    hipMemcpy(d_flag,h_flag,sizeof(int)*len,hipMemcpyHostToDevice);
    delete[] h_flag;

    int res = nth_order_stat((int)((len+0.5)/2), arr, d_arr, d_flag, len);
    
    hipFree(d_arr);
    hipFree(d_flag);
    return res;
}

int main()
{
    //开始时间戳
    clock_t h_t1 = clock();

    //读入数据
    vector<string> files;
    scan_dir("./data40M", files);
    vector<int> h_vec; 
    read_data(files,h_vec);

    //读取数据时间戳
    clock_t h_t2 = clock();

    //GPU运算开始时间
    hipEvent_t     d_start, d_stop;
    hipEventCreate( &d_start );
    hipEventCreate( &d_stop );
    hipEventRecord( d_start, 0 );

    //调用算法
    int res = median(h_vec.data(), h_vec.size());

    //GPU运算结束时间
    hipEventRecord( d_stop, 0 );
    hipEventSynchronize( d_stop );
    float   elapsedTime;
    hipEventElapsedTime( &elapsedTime, d_start, d_stop );
    printf( "Time on GPU:  %.3f s\n", elapsedTime/1000 );

    //结束时间戳
    clock_t h_end = clock();
    cout<<"Median: "<<res<<endl;
    cout<<"IO Time: "<<(double)(h_t2-h_t1)/CLOCKS_PER_SEC<<"s"<<endl;
    cout<<"Algorithm Time: "<<(double)(h_end-h_t2)/CLOCKS_PER_SEC<<"s"<<endl;
    cout<<"Total Time: "<<(double)(h_end-h_t1)/CLOCKS_PER_SEC<<"s"<<endl;

    hipEventDestroy( d_start );
    hipEventDestroy( d_stop );
    return 0;
}
